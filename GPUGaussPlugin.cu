#include <emmintrin.h>
#include <sys/time.h> 
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <iostream>
#include <iomanip>
#include <fstream>

#include "GPUGaussPlugin.h"

void GPUGaussPlugin::input(std::string file) {
 inputfile = file;
 std::ifstream ifile(inputfile.c_str(), std::ios::in);
 while (!ifile.eof()) {
   std::string key, value;
   ifile >> key;
   ifile >> value;
   parameters[key] = value;
 }
 N = atoi(parameters["N"].c_str());
 a = (float*) malloc(N*N*sizeof(float));
 int M = N * N;
 std::ifstream myinput((std::string(PluginManager::prefix())+parameters["matrix"]).c_str(), std::ios::in);
 int i;
 for (i = 0; i < M; ++i) {
	int k;
	myinput >> k;
        a[i] = k;
 }
}

void GPUGaussPlugin::run() {
    int i;
    for (i = 32; i <= 1024; i += 32){
        if (i >= N) { break; }
    }
    int numThreads = i;
    int numCores = numThreads;

    float* gpuA;

    // Allocate enough memory on the GPU
    hipMalloc(&gpuA, N * N * sizeof (float)); 

    // Copy array from CPU to GPU
    hipMemcpy(gpuA, a, N * N * sizeof (float), hipMemcpyHostToDevice); 


    for (i = 0; i < N; i++){
        gpu_zeroColumn <<<numCores, numThreads>>>(gpuA, N, i); 
    }

    // Copy array from GPU to CPU
    hipMemcpy(a, gpuA, N * N * sizeof (float), hipMemcpyDeviceToHost); 

    // Free the memory on the GPU
    hipFree(&gpuA); 
}

void GPUGaussPlugin::output(std::string file) {
	std::ofstream outfile(file.c_str(), std::ios::out);
        int i, j;
        for (i = 0; i < N; ++i){
            for (j = 0; j < N; ++j){
		outfile << (int) a[i*N+j];//std::setprecision(0) << a[i*N+j];
		if (j != N-1)
			outfile << "\t";
		else
			outfile << "\n";
            }
	}
	free(a);
}



PluginProxy<GPUGaussPlugin> GPUGaussPluginProxy = PluginProxy<GPUGaussPlugin>("GPUGauss", PluginManager::getInstance());


